#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    /* insira codigo para calcular o indice usando blockIdx.x, blockDim.x, threadIdx.x */
	//int index = threadIdx.x;/* RESPONDA */
	int index = blockIdx.x;
	c[index] = a[index] + b[index];
}

/* EXPERIMENTE com N */
/* o quao grande ele pode ser? */
#define N (32*32)
#define THREADS_PER_BLOCK 512

int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	int size = N * sizeof( int );

	/* aloca espaco no device para copias de a, b, c */
	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* aloca espaco no host para copias de a, b, c e inicializa valores */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	/* copia dados para o device */
	/* COMPLETE com os parametros necessários para copiar dados para o device */
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	/* lance o kernel na GPU */
	/* COMPLETE com os parametros para lançar o kernel propriamente usando blocos e threads */
	add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOC>>>( d_a, d_b, d_c );

	/* copie os resultados de volta ao host */
	/* COMPLETE com parametros necessarios para copiar os dados de volta ao host */
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost );


	printf( "c[0] = %d\n", c[0] );
	printf( "c[%d] = %d\n",N-1, c[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
