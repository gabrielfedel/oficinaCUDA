
#include <hip/hip_runtime.h>
#include <stdio.h>


// indica que é uma fun��o que vai rodar no device
__global__ void hello()
{
	printf("Oi mundo! De thread: %d De: bloco %d\n", threadIdx.x, blockIdx.x);
}

int main(void)
{
	int num_threads = 5;
	int num_blocks = 5;
	//chama a fun��o e espec�fica blocos e threads
	hello<<<num_blocks,num_threads>>>();
	//espera o c�digo da gpu terminar
	hipDeviceSynchronize();

	return 0;
}
